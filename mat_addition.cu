
#include <hip/hip_runtime.h>
#include<iostream>
#include<string>

int main(int argc, char* argv[])
{
	int N;
	if(argc == 2)
	{
		N = std::stoi(argv[1]);
	}
	else
	{
		std::cerr << "Usage: ./mat_add N"<< "\n";
		return -1;
	}

	float* A = new float[N * N];
	float* B = new float[N * N];
	float* C = new float[N * N];

	for(int i=0; i<N; i++)
	{
		for(int j=0; j<N; j++)
		{
			A[i * N + j]=1.0;
			B[i * N + j]=1.0;
			C[i * N + j]=0.0;
		}
	}

	for(int i=0; i<N; i++)
	{
		for(int j=0; j<N; j++)
		{
			C[i * N + j]= A[i * N + j] + B[i * N + j];
		}
	}


	for(int i=0; i<N; i++)
	{
		for(int j=0; j<N; j++)
		{
			std::cout << C[i * N + j] << " ";
		}
		std::cout << "\n";
	}

	delete[] A;
	delete[] B;
	delete[] C;

	return 0;
}
